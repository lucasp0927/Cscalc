#include "matrixMul.h"
// void deviceVerify ()
// {
//   int devID;
//   hipDeviceProp_t props;
//   // get number of SMs on this GPU
//   hipGetDevice(&devID);
//   hipGetDeviceProperties(&props, devID);
//   printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name, props.major, props.minor);
// }

void zdot(double* A,double* B,int N,double* result)
{
  unsigned int mem_size = sizeof(hipDoubleComplex)*N*N;
  hipDoubleComplex* d_result,* d_A,* d_B;
  hipMalloc((void**) &d_result,mem_size);
  hipMalloc((void**) &d_A,mem_size);
  hipMalloc((void**) &d_B,mem_size);
  hipMemcpy(d_A, A, mem_size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, mem_size, hipMemcpyHostToDevice);  
  const hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);
  const hipDoubleComplex beta = make_hipDoubleComplex(0.0,0.0);
  hipEventRecord(start, 0);

  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_B, N, d_A, N, &beta, d_result, N);
  hipMemcpy(result, d_result, mem_size, hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_result);  
}

// void cdot(float* A,float* B,int N,float* result)
// {
//   hipEvent_t start, stop;
//   float time;
//   hipEventCreate(&start);
//   hipEventCreate(&stop);
  
//   hipblasHandle_t handle;
//   hipblasCreate(&handle);
// //checkError(hipblasCreate(&handle), "hipblasCreate() error!\n");  
//   unsigned int mem_size = sizeof(hipComplex)*N*N;
  
//   hipComplex* d_result,* d_A,* d_B;
//   hipMalloc((void**) &d_result,mem_size);
//   hipMalloc((void**) &d_A,mem_size);
//   hipMalloc((void**) &d_B,mem_size);
//   hipMemcpy(d_A, A, mem_size, hipMemcpyHostToDevice);
//   hipMemcpy(d_B, B, mem_size, hipMemcpyHostToDevice);  
//   const hipComplex alpha = make_hipComplex(1.0,0.0);
//   const hipComplex beta = make_hipComplex(0.0,0.0);
//   hipEventRecord(start, 0);

//   hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_B, N, d_A, N, &beta, d_result, N);
//   hipEventRecord(stop, 0);
//   hipEventSynchronize(stop);
//   hipEventElapsedTime(&time, start, stop);
//   printf ("Time for the hipblasZgemm: %f ms\n", time);
  
//   hipMemcpy(result, d_result, mem_size, hipMemcpyDeviceToHost);
  
//   hipFree(d_A);
//   hipFree(d_B);
//   hipFree(d_result);  
// }

